

/**
 * Matrix multiplication: C = A * B.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling approach.
 */

// System includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <assert.h>

#include <sys/time.h>
#include <unistd.h>
#define TILESIZE 32
/**
 * Matrix multiplication : C = A * B
 * wA is A's width and wB is B's width
 */
void MatrixMul(float *C, float *A, float *B, int hA, int wA, int wB) {
	for (int row = 0; row < hA; row++){
		for (int col = 0; col < wB; col++){
			C[row * wA + col] = 0;
			for (int k = 0; k < wA; k++){
				C[row * wA + col] += A[row * wA + k] * B[k * wB + col];
			}
		}
	}
}

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 * CUDA Kernel 1: each thread work independently on one element in C
 */
__global__
//<template int TILESIZE>
void MatrixMulCuda2(float *d_C, float *d_A, float *d_B, int hA, int wA, int wB, int tile_size) {
	//TODO
	// tile_size = block_size
	//const int TILE_SIZE = tile_size;
	int height = hA, width = wB;
//	const int TILE_SIZE = tile_size;
//	printf("The tile size is: %d",tile_size);
//	__shared__  float d_AS [32][32];
//	__shared__  float d_BS [32][32];
	__shared__  float d_AS [TILESIZE][TILESIZE];
	__shared__  float d_BS [TILESIZE][TILESIZE];
	int tx,ty;
	ty = threadIdx.y;
	tx = threadIdx.x;
	int col = tx + blockDim.x*blockIdx.x;
	int row = ty + blockDim.y*blockIdx.y;
	int phase_size = (((hA>wA?(hA>wB?hA:wB):(wA>wB?wA:wB))) + tile_size -1)/tile_size;
	//printf("%d,%d,%d,%d\n",hA,wA,wB,phase_size);
	float p_val = 0;
	if (row<=wB && col<=hA){
		for (int ph=0;ph<phase_size;++ph){

		d_AS[ty][tx] = d_A[row*wA+ph*tile_size+tx];
		d_BS[ty][tx] = d_B[(ph*tile_size+ty)*wB+col];
		__syncthreads();
		for (int k=0;k<tile_size;++k){
			p_val += d_AS[ty][k]*d_BS[k][tx];
		}
		__syncthreads();

//		p_val = 0;
		//__syncthreads();
		}
		d_C[row*wB+col] = p_val;
	}

}

__global__ void MatrixMulCuda1(float *d_C, float *d_A, float *d_B, int hA, int wA, int wB, int tile_size) {

	__shared__  float ds_M [TILESIZE][TILESIZE];
	__shared__  float ds_N [TILESIZE][TILESIZE];
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Row = by*blockDim.y+ty;
	int Col = bx*blockDim.x+tx;
	int phase_size = (((hA>wA?(hA>wB?hA:wB):(wA>wB?wA:wB))) + tile_size -1)/tile_size;
	float Pvalue = 0;
	for (int p=0;p<phase_size;++p){
		ds_M[ty][tx] = d_A[Row*wA + p*tile_size+tx];
		ds_N[ty][tx] = d_B[(p*tile_size+ty)*wA+Col];

	__syncthreads();
	for (int k=0;k<tile_size;++k){
		Pvalue += ds_M[ty][k]*ds_N[k][tx];
	}
	__syncthreads();
	d_C[Row*wB+Col] = Pvalue;
	}


}
/**
 * Initialize the array
 */
void ConstantInit(float *data, int size, float val) {
    for (int i = 0; i < size; ++i) {
        data[i] = val;
    }
}

/**
 * Evaluate whether the CUDA method is successfully executed
 */
void checkCudaErrors(hipError_t  err){
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Evaluate whether the CUDA method is successfully executed
 */
void evaluatePerformance(double msecTotal, int nIter, double nFloatOps, int threadsIdx_x, int threadsIdx_y){
	// Compute and print the performance
	float msecPerMatrixMul = msecTotal / nIter;
	double flopsPerMatrixMul = nFloatOps;
	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) /
					   (msecPerMatrixMul / 1000.0f);
	printf(
		"Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops," \
		" WorkgroupSize= %u threads/block\n",
		gigaFlops,
		msecPerMatrixMul,
		flopsPerMatrixMul,
		threadsIdx_x * threadsIdx_y);
}

/**
 * Run a simple test of matrix multiplication
 */
int MatrixMultiply(int hA, int wA, int wB) {

    // Allocate host memory for matrices A and B
    unsigned int size_A = hA * wA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)(malloc(mem_size_A));
    unsigned int size_B = wA * wB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)(malloc(mem_size_B));
    int block_size = 2;
    int tile_size = 2;
    // Initialize host memory
    const float valB = 0.01f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate host matrix C
    unsigned int mem_size_C = hA * wB * sizeof(float);
    float *h_C = (float *)(malloc(mem_size_C));

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate matrix C!\n");
        exit(-1);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));

    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));

    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));

    // Setup execution parameters

    dim3 threads(block_size, block_size);
    dim3 grid((wB + block_size - 1)/ block_size, (hA + block_size - 1) / block_size);
    MatrixMulCuda2<<<grid, threads>>>(d_C, d_A, d_B, hA, wA, wB, tile_size);
    hipDeviceSynchronize();
    /*
	struct timeval begin, end;
	gettimeofday(&begin, NULL);

    MatrixMulCuda1<<<grid, threads>>>(d_C, d_A, d_B, hA, wA, wB);
	cudaDeviceSynchronize();
	gettimeofday(&end, NULL);

	long seconds = end.tv_sec - begin.tv_sec;
	long micros = (seconds * 1000000) + end.tv_usec - begin.tv_usec;
	*/

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    evaluatePerformance(msecTotal, 1, (double)(2 * hA * wA * wB), block_size, block_size);

    // Copy result from device to host
    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));


    printf("done\n");

    printf("Checking computed result for correctness: \n");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6;  // machine zero

    for (int i = 0; i < (int)(hA * wB); i++) {
        double abs_err = abs(h_C[i] - (wA * valB));
        double dot_length = wA;
        double abs_val = abs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;

        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                   i, h_C[i], wA * valB, eps);
            correct = false;

        }
    }
//    for (int i = 0; i < (int)(hA * wB); i++) {
//    	printf("%d,%f\n",i,h_C[i]);
//    }
//    correct = false;
    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);

    if (correct) {
        return 0;
    } else {
        return -1;
    }
}


/**
 * Program main
 */
int main(int argc, char **argv) {
    printf("[Matrix Multiply Using C] - Starting...\n");

//    if (argc < 4){
//        printf("Usage -device=n (n >= 0 for deviceID)\n");
//        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
//        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
//        printf("  Note: Outer matrix dimensions of A & B matrices" \
//               " must be equal.\n");
//
//        exit(EXIT_SUCCESS);
//    }
//
//    int hA = atoi(argv[1]);
//    int wA = atoi(argv[2]);
//    int wB = atoi(argv[3]);
//    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", hA, wA,

    //test case: tile_size = 2, input = 5
    int hA = 100;
    int wA = 100;
    int wB = 100;
//    int hA = 5;
//	int wA = 5;
//	int wB = 5;
    MatrixMultiply(hA, wA, wB);

//	printf("Time elapsed is %ld seconds and %ld micros\n", seconds, micros);

//    exit(matrix_result);
}

