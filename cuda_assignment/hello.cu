
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void hello(void){
	printf("Hello from the GPU\n");
}

int main(void){
	printf("Hello from the CPU\n");
	dim3 gridDim(2,3,1);
	// setup 6 blocks
	dim3 blockDim(10,2,3);
	// setup 6 blocks, a diff setup;
	hello<<<gridDim,blockDim>>>();
	hipDeviceReset();
	return 0;
}
