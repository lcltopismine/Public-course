#include "hip/hip_runtime.h"
/**
*	Matrix multiplication: C = A * B.
*	
*	This sample implements matrix multiplication which makes use of shared memory
*	to ensure data reuse, the matrix multiplication is done using tiling approach.
*/

//	System includes 
#include <stdio.h> 
#include <stdlib.h> 
#include <stdbool.h> 
#include <assert.h>
#include <sys/time.h>
#include <unistd.h>

/**
*	Matrix multiplication : C = A * B
*	wA is A's width and wB is B's width
*/

void MatrixMul(float *C, float *A, float *B, int hA, int wA, int wB) { 
	for (int row = 0; row < hA; row++){
		for (int col = 0; col < wB; col++){
			C[row * wA + col] = 0;
			for (int k = 0; k < wA; k++){
				C[row * wA + col] += A[row * wA + k] * B[k * wB + col];
			}
		}
	}
}

/**
*	Matrix multiplication (CUDA Kernel) on the device: C = A * B
*	wA is A's width and wB is B's width
*	CUDA Kernel 1: each thread work independently on one element in C
*/
__global__
void MatrixMulCuda1(float *d_C, float *d_A, float *d_B, int hA, int wA, int wB) { 
	const int tile_size = 512;
	__shared__ float ds_M[tile_size][tile_size];
	__shared__ float ds_N[tile_size][tile_size];
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.y*blockDim.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	double p_val = 0;
	// assume divisible
	int num_phase = hA/tile_size;
	if (row<hA and col<wB){
		for(int i=0;i<num_phase;i++){
			ds_M[ty][tx] = d_A[row*width+i*tile_size+tx];
			ds_N[ty][tx] = d_B[(i*tile_size+ty)*width+col];
			__syncthreads();
			for(k=0;k<tile_size;k++){
				p_val += ds_M[ty][k] * ds_N[k][tx];
			}
			__syncthreads();
		}
		d_C[row*wA+col] = p_val;
	}
	
}

/**
*	Initialize the array
*/
void ConstantInit(float *data, int size, float val) {
	for (int i = 0; i < size; ++i) {
		data[i] = val;
	}
}

/**
*	Evaluate whether the CUDA method is successfully executed
*/
void checkCudaErrors(hipError_t err){
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

/**
*	Evaluate whether the CUDA method is successfully executed
*/
void evaluatePerformance(double msecTotal, int nIter, double nFloatOps, int threadsIdx_x, int threadsIdx_y){
	// Compute and print the performance
	float msecPerMatrixMul = msecTotal / nIter;
	double flopsPerMatrixMul = nFloatOps;
	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
	printf(
		"Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops," \
		"	WorkgroupSize= %u threads/block\n", gigaFlops,
		msecPerMatrixMul,
		flopsPerMatrixMul, threadsIdx_x * threadsIdx_y);
}

/**
*	Run a simple test of matrix multiplication
*/
int MatrixMultiply(int hA, int wA, int wB) {
	//	Allocate host memory for matrices A and B 
	unsigned int size_A = hA * wA;
	unsigned int mem_size_A = sizeof(float) * size_A; 
	float *h_A = (float *)(malloc(mem_size_A));
	unsigned int size_B = wA * wB;
	unsigned int mem_size_B = sizeof(float) * size_B; 
	float *h_B = (float *)(malloc(mem_size_B));

	//	Initialize host memory 
	const float valB = 0.01f; 
	ConstantInit(h_A, size_A, 1.0f); 
	ConstantInit(h_B, size_B, valB);

	//	Allocate host matrix C
	unsigned int mem_size_C = hA * wB * sizeof(float); 
	float *h_C = (float *)(malloc(mem_size_C));
	if (h_C == NULL) {
		fprintf(stderr, "Failed to allocate matrix C!\n");
		exit(-1);
	}

	//	Allocate device memory 
	float *d_A, *d_B, *d_C;
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

	// copy host memory to device
	checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

	//	Allocate CUDA events that we'll use for timing 
	hipEvent_t start; 
	checkCudaErrors(hipEventCreate(&start));

	hipEvent_t stop;
	checkCudaErrors(hipEventCreate(&stop));

	//	Record the start event 
	checkCudaErrors(hipEventRecord(start, NULL));

	//	Setup execution parameters
	int block_size = 4;
	dim3 threads(block_size, block_size);
	dim3 grid((wB + block_size - 1)/ block_size, (hA + block_size - 1) / block_size); 
	MatrixMulCuda1<<<grid, threads>>>(d_C, d_A, d_B, hA, wA, wB);

	/*
	struct timeval begin, end;
	gettimeofday(&begin, NULL);
	MatrixMulCuda1<<<grid, threads>>>(d_C, d_A, d_B, hA, wA, wB); 
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);
	long seconds = end.tv_sec - begin.tv_sec;
	long micros = (seconds * 1000000) + end.tv_usec - begin.tv_usec; 
	*/

	//	Record the stop event 
	checkCudaErrors(hipEventRecord(stop, NULL));

	//	Wait for the stop event to complete 
	checkCudaErrors(hipEventSynchronize(stop));

	float msecTotal = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
	evaluatePerformance(msecTotal, 1, (double)(2 * hA * wA * wB), block_size, block_size);

	// Copy result from device to host
	checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

	printf("done\n");
	printf("Checking computed result for correctness: "); bool correct = true;

	//	test relative error by the formula
	//	|<x, y>_cpu - <x,y>_gpu|/<|x|, |y|> < eps double eps = 1.e-6; // machine zero

	for (int i = 0; i < (int)(hA * wB); i++) {
		double abs_err = abs(h_C[i] - (wA * valB));
		double dot_length = wA;
		double abs_val = abs(h_C[i]);
		double rel_err = abs_err / abs_val / dot_length;
		if (rel_err > eps) {
			printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], wA * valB, eps);
			correct = false;
		}
	}

	printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

	//	Clean up memory 
	free(h_A); 
	free(h_B); 
	free(h_C); 
	hipFree(d_A); 
	hipFree(d_B); 
	hipFree(d_C);

	if (correct) {
		return 0;
	} else { 
		return -1;
	}
}

/**
*	Program main
*/
int main(int argc, char **argv) {
	printf("[Matrix Multiply Using C] - Starting...\n");

	int hA = 100;
	int wA = 100; 
	int wB = 100;
	printf(" -wA=WidthA:%d -hA=HeightA:%d (Width x Height of Matrix A)\n", wA, hA); 
	printf(" -wB=WidthB:%d -hB=HeightB:%d (Width x Height of Matrix B)\n", wB, wA);

	MatrixMultiply(hA, wA, wB);
}
