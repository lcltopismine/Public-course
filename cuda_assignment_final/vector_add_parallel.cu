#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define N 10000000
#define MAX_ERR 1e-6

__global__
void vector_add(float *out, float *a, float *b, int n) {
	// this is the global function
	// transfer memory into device
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index<n)
        out[index] = a[index] + b[index];
}

int main(){
    float *a, *b, *out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Main function
    int thread_per_grid = 256;
    dim3 gridDim(thread_per_grid,1,1);
    dim3 blockDim((n-1)/thread_per_grid+1,1,1);
	float *d_A,*d_B;
	int size = N*sizeof(float);
	hipMalloc((void **) &d_A,size);
	hipMalloc((void **) &d_B,size);
	hipMalloc((void **) &d_out,size);
	hipMemcpy(d_A, a, size,hipMemcpyHostToDevice);
	hipMemcpy(d_B, b, size,hipMemcpyHostToDevice);
    vector_add<<<gridDim,blockDim>>>(d_out, d_A, d_B, N);
    hipMemcpy(out, d_out, size,hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_out);
    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");
}
