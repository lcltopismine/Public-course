
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define N 10000000
#define MAX_ERR 1e-6

__global__
void vector_add(float *out, float *a, float *b, int n) {
	int i = blockDim.x*blockIdx.x+threadIdx.x;
    if (i<n)
        out[i] = a[i] + b[i];

}

int main(){
    float *a, *b, *out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Main function
    int block_size=256;
    dim3 gridDim((N-1)/block_size+1,1,1);
    dim3 blockDim(block_size,1,1);
    //transfer memory
    float *d_A,*d_B,*d_out;
    int size = sizeof(float) * N;
    hipMalloc((void **)&d_A,size);
    hipMalloc((void **)&d_B,size);
    hipMalloc((void **)&d_out,size);
    hipMemcpy(d_A,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,b,size,hipMemcpyHostToDevice);
    vector_add<<<gridDim,blockDim>>>(d_out, d_A, d_B, N);
    hipMemcpy(out,d_out,size,hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_out);

    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");
}
