#include "hip/hip_runtime.h"
/*
Convert an image to grayscale.

The code is written for a mini-project of ITCS 5145 Parallel Programming at UNCC.(Q.C.)

To compile the code, we use
        gcc -g -Wall -o color2grapy stb_image/stb_image.h stb_image/stb_image_write.h color2gray.c -lm

To run the code, type
        ./color2gray ${input color image} ${output grayscale image} ${image type}

        The format of images depends on its types.
        To specify image type, we have ${image type} as follows:
            1 is for .png file
            2 is for .jpg file
        
        For example,
        ./color2grapy lena1.png lena2.png 1
        ./color2grapy lizard1.jpg lizard2.jpg 2
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"

__global__
void colorToGrayscale(unsigned char *gray_img, unsigned char * color_img, int width, int height);

const int IS_PNG = 1;
const int IS_JPG = 2;
const int DESIRED_CHANNELS = 3;
const int MAX_NAME_LENGTH = 500;

int main(int argc, char *argv[]) {
	argv[1] = "//home//student//Desktop//cuda_image//lena1.png";
	argv[2] = "//home//student//Desktop//cuda_image//out.png";
	argv[3] = "1";

    int width, height, channels, type;
    char  in_name[MAX_NAME_LENGTH], out_name[MAX_NAME_LENGTH];
    strcpy(in_name, argv[1]);
    strcpy(out_name, argv[2]);
    type = atoi(argv[3]);
    printf("name: %s\n",in_name);
    unsigned char *color_img = stbi_load(in_name, &width, &height, &channels, 0); // load and conver the image to 3 channels (ignore the transparancy channel)
    if(color_img == NULL) {
        printf("Error in loading the image\n");
        exit(-1);
    }
    printf("Loaded image %s with a width of %dpx, a height of %dpx and %d channels\n", in_name, width, height, channels);

    // Convert the input image to gray
    int gray_channels = channels == 4 ? 2 : 1;
    size_t gray_img_size = width * height * gray_channels;
    size_t color_img_size = width * height * DESIRED_CHANNELS;
    
    unsigned char *gray_img = (unsigned char *)malloc(gray_img_size);
    if(gray_img == NULL) {
        printf("Unable to allocate memory for the gray image.\n");
        exit(1);
    }
    printf("Create a image array with a width of %dpx, a height of %dpx and %d channels\n", width, height, gray_channels);

    //setup device side memory
    unsigned char *d_color_img, *d_gray_img;
    hipMalloc((void **) &d_color_img, color_img_size);
    hipMalloc((void **) &d_gray_img, gray_img_size);
    hipMemcpy(d_color_img, color_img, color_img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_gray_img, gray_img, gray_img_size, hipMemcpyHostToDevice);
    //kernel function starts
    int block_size = 16;
    int per_block = (height-1)/block_size +1;
    dim3 DimGrid(per_block,per_block,1);
    dim3 DimBlock(block_size,block_size,1);
    printf("per_block: %d\n",per_block);
    colorToGrayscale<<<DimGrid,DimBlock>>>(d_gray_img, d_color_img, width, height);
    //kernel function ends
    hipMemcpy(gray_img, d_gray_img, gray_img_size, hipMemcpyDeviceToHost);
    hipFree(d_color_img);
    hipFree(d_gray_img);


    if (type == IS_PNG)
    	stbi_write_png(out_name, width, height, gray_channels, gray_img, width * gray_channels);
    else
        if (type == IS_JPG)
            stbi_write_jpg(out_name, width, height, gray_channels, gray_img, 100); //The last parameter of the stbi_write_jpg function is a quality parameter that goes from 1 to 100. Since JPG is a lossy image format, you can chose how much data is dropped at save time. Lower quality means smaller image size on disk and lower visual image quality.
    printf("Wrote image %s with a width of %dpx, a height of %dpx and %d channels\n", out_name, width, height, channels);

    stbi_image_free(gray_img); 
}

__global__
void colorToGrayscale(unsigned char *gray_img, unsigned char * color_img, int width, int height){
    unsigned char pixel[DESIRED_CHANNELS];

   //If the input image has a transparency channel this will be simply copied to the second channel of the gray image, while the first channel of the gray image will contain the gray pixel values. If the input image has three channels, the output image will have only one channel with the gray data.
    int row = threadIdx.x + blockDim.x*blockIdx.x;
    int col = threadIdx.y + blockDim.y*blockIdx.y;
//    printf("%d,%d\n",row,col);
    if (row<width && col<height){
		int greyOffset = col * width + row;
		//printf("%d\n",greyOffset);
		int rgbOffset = greyOffset * DESIRED_CHANNELS;
		pixel[0] = color_img[rgbOffset];
		pixel[1] = color_img[rgbOffset + 1];
		pixel[2] = color_img[rgbOffset + 2];

		gray_img[greyOffset] = pixel[0] * 0.3 + pixel[1] * 0.58 + pixel[2] * 0.11;
    }
}
