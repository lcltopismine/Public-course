#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define N 10000000
#define MAX_ERR 1e-6


__global__
void vector_add_kernel(float *out, float *a, float *b, int n){
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    //printf("%d,%d,%d\n",i,n,a[i]);
    if (i<n)
    	out[i] = a[i] + b[i];
    //printf("%d!",out[i]);
}

__host__
void vector_add(float *out, float *a, float *b, int n){
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_out;
    hipMalloc((void **) &d_A, size);
    hipMemcpy(d_A, a, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, b, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_out, size);

    int block_size = 256;
    int total_block = (N + block_size -1)/block_size;
    //printf("%f,%f\n",a[0],b[0]);
    //printf("%d,%d", total_block,block_size);
    vector_add_kernel<<<total_block,block_size>>>(d_out, d_A, d_B, N);

    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_out);
}
int main(){
    float *a, *b, *out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Main function
    vector_add(out, a, b, N);

    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");
}
